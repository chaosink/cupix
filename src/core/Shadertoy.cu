#include "hip/hip_runtime.h"
#include "glm/glm.hpp"

namespace cupix {

namespace kernel {

extern __constant__ __device__ int w, h;
extern __constant__ __device__ float time;

#define iTime time
#define iResolution vec2(w, h)

using namespace glm;

__device__ // https://www.shadertoy.com/view/Xt2czt
vec4 FlickeringDots(vec2 fragCoord) {
	vec2 uv = fragCoord - iResolution / 2.f;
	float d = dot(uv, uv);
	return vec4(0.5f + 0.5f * cos(d / 5.f + 10.f * time));
}

__device__ // https://www.shadertoy.com/view/lljSDy
vec4 Quadtree(vec2 U) {
	vec4 o;
	o -= o;
	float r=.1f, t=time, H = h;
	U /=  H;							// object : disc(P,r)
	vec2 P = .5f+.5f*vec2(cos(t),sin(t*.7f)), fU;
	U*=.5f; P*=.5f;						// unzoom for the whole domain falls within [0,1]^n
	o.b = .25f;							// backgroud = cold blue
	for (int i=0; i<7; i++) {			// to the infinity, and beyond ! :-)
		fU = min(U,1.f-U); if (min(fU.x,fU.y) < 3.f*r/H) { o--; break; } // cell border
		if (length(P-.5f) - r > .7f) break; // cell is out of the shape
				// --- iterate to child cell
		fU = step(.5f,U);				// select child
		U = 2.f*U - fU;					// go to new local frame
		P = 2.f*P - fU;  r *= 2.f;

		o += .13f;						// getting closer, getting hotter
	}
	o.g *= smoothstep(.9f,1.f,length(P-U)/r); // draw object
	o.b *= smoothstep(.9f,1.f,length(P-U)/r);
	return o;
}

#define N 10.f
__device__ // https://www.shadertoy.com/view/4sjSRt
vec4 Sunflower(vec2 u) {
	vec4 o;
	o.x = w; o.y = h;
	u = (u+u-vec2(w,h))/o.y;
	//u = 2.*(u / iResolution.y -vec2(.9,.5));
	float t = time,
		r = length(u), a = atan(u.y,u.x),
		i = floor(r*N);
	a *= floor(pow(128.f,i/N)); 	 a += 20.f*sin(.5f*t)+123.34f*i-100.f*r*cos(.5f*t); // (r-0.*i/N)
	r +=  (.5f+.5f*cos(a)) / N;	r = floor(N*r)/N;
	o = (1.f-r)*vec4(.5f,1.f,1.5f,1.f);
	return o;
}
#undef N

//#define BALLS
__device__ // https://www.shadertoy.com/view/4dsSzS
vec4 Mandeltunnel(vec2 fragCoord) {
	vec2 uv = -1.f + 2.f*fragCoord / iResolution;
	uv.x *= iResolution.x/iResolution.y;

	vec2 c = vec2(-uv.y-0.3f, uv.x)*0.13f;
	vec2 z = vec2(0.f);

	float sinTime = sin(iTime*1.5f);
	vec2 cinc = vec2(sinTime*0.0001f, cos(iTime)*0.0003f);
	float m = clamp((abs(uv.x)+uv.y+0.25f)*2.f,-1.f,1.f);

	float ni = 0.f;
	vec4 col = vec4(0.f,0.f,0.f,1.f);
	bool hit = false;
	for(float i=0.f; i<50.f; i++) {
		if (hit) continue;

		float f = sin(time);
		z = vec2(z.x*z.x - z.y*z.y, 2.f*z.x*z.y) + i*c;
		float r = dot(z,z);

		if(r > 1.3f+f*m - sinTime*0.1f) {
			hit = true;
			ni = 1.f - ni;

			col = vec4(1.f+f,ni,ni,1.f)*ni;

			#ifdef BALLS
				col *= 0.5f*r-ni*0.75f;
			#endif

		}
		ni += 0.02f;
		c += cinc;
	}
	return col;
}

#define MAX_ITERS 150.f
__device__ // https://www.shadertoy.com/view/Mdj3Rh
vec4 MandelbrotsDarkerSide(vec2 fragCoord) {
	vec2 uv = fragCoord / iResolution;
	vec2 c = (2.f * uv - 1.f)
			 * vec2(iResolution.x / iResolution.y, 1.f);
	// view
	c.x -=.3f;
	c *= 1.5f;

	vec2 z;

	float iters = 20.f*(1.f-cos(((.2f*iTime*6.f+6.f*log(.5f*iTime*6.f+1.f))*.9f)*.05f));

	for (float i = 0.f; i < MAX_ITERS; ++i) {
		if( i > iters ) continue;
		float alpha = clamp(iters-float(i),0.f,1.f);
		alpha = smoothstep(0.f,1.f,alpha);
		vec2 newz = vec2(z.x  * z.x - z.y * z.y, 2.f * z.x * z.y ) + c;
		// simple linear interpolation
		z = (1.f-alpha)*z + alpha*newz;
	}

	float col = (z.x*z.x+z.y*z.y);

	col = pow(col,.35f);
	col = clamp(col,0.f,1.1f);

	float vign = (1.f-.5f*dot(uv-.5f,uv-.5f));
	return vec4(vec3(.95f,.95f,.8f)*(col) * vign, 1.f);
}
#undef MAX_ITERS

__device__ // https://www.shadertoy.com/view/4dX3Rn
vec4 DeformFlower(vec2 fragCoord) {
	vec2 p = (2.0f*fragCoord-iResolution)/min(iResolution.y,iResolution.x);

	float a = atan(p.x,p.y);
	float r = length(p)*(0.8f+0.2f*sin(0.3f*iTime));

	float w = cos(2.0f*iTime-r*2.0f);
	float h = 0.5f+0.5f*cos(12.0f*a-w*7.0f+r*8.0f+ 0.7f*iTime);
	float d = 0.25f+0.75f*pow(h,1.0f*r)*(0.7f+0.3f*w);

	float f = sqrt(1.0f-r/d)*r*2.5f;
	f *= 1.25f+0.25f*cos((12.0f*a-w*7.0f+r*8.0f)/2.0f);
	f *= 1.0f - 0.35f*(0.5f+0.5f*sin(r*30.0f))*(0.5f+0.5f*cos(12.0f*a-w*7.0f+r*8.0f));

	vec3 col = vec3( f,
					 f-h*0.5f+r*.2f + 0.35f*h*(1.0f-r),
					 f-h*r + 0.1f*h*(1.0f-r) );
	col = clamp( col, 0.0f, 1.0f );

	vec3 bcol = mix( 0.5f*vec3(0.8f,0.9f,1.0f), vec3(1.0f), 0.5f+0.5f*p.y );
	col = mix( col, bcol, smoothstep(-0.3f,0.6f,r-d) );

	return vec4( col, 1.0f );
}

__device__ // https://www.shadertoy.com/view/XsfGRn
vec4 Heart2D(vec2 fragCoord) {
	vec2 p = (2.0f*fragCoord-iResolution)/min(iResolution.y,iResolution.x);

	// background color
	vec3 bcol = vec3(1.0f,0.8f,0.7f-0.07f*p.y)*(1.0f-0.25f*length(p));

	// animate
	float tt = mod(iTime,1.5f)/1.5f;
	float ss = pow(tt,.2f)*0.5f + 0.5f;
	ss = 1.0f + ss*0.5f*sin(tt*6.2831f*3.0f + p.y*0.5f)*exp(-tt*4.0f);
	p *= vec2(0.5f,1.5f) + ss*vec2(0.5f,-0.5f);

	// shape
#if 0
	p *= 0.8f;
	p.y = -0.1f - p.y*1.2f + abs(p.x)*(1.0f-abs(p.x));
	float r = length(p);
	float d = 0.5f;
#else
	p.y -= 0.25f;
	float a = atan(p.x,p.y)/3.141593f;
	float r = length(p);
	float h = abs(a);
	float d = (13.0f*h - 22.0f*h*h + 10.0f*h*h*h)/(6.0f-5.0f*h);
#endif

	// color
	float s = 0.75f + 0.75f*p.x;
	s *= 1.0f-0.4f*r;
	s = 0.3f + 0.7f*s;
	s *= 0.5f+0.5f*pow( 1.0f-clamp(r/d, 0.0f, 1.0f ), 0.1f );
	vec3 hcol = vec3(1.0f,0.5f*r,0.3f)*s;

	vec3 col = mix( bcol, hcol, smoothstep( -0.01f, 0.01f, d-r) );

	return vec4(col,1.0f);
}

}
}
