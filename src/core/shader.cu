#include "hip/hip_runtime.h"
#include "cupix.hpp"

namespace cupix {

namespace cu {

extern __constant__ __device__ int w, h;
extern texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> texture;

__device__
void VertexShader(VertexIn &in, VertexOut &out, glm::mat4 &mvp) {
	out.position = mvp * glm::vec4(in.position, 1.f);
	out.normal = in.normal;
	out.color = in.color;
	out.uv = in.uv;
}

__device__
void FragmentShader(FragmentIn &in, glm::vec4 &color) {
	// glm::vec4 c = glm::vec4(in.normal, 0.f);
	glm::vec4 c = glm::vec4(in.normal * 0.5f + 0.5f, 0.f);
	// glm::vec4 c = glm::vec4(in.uv, 0.f, 0.f);
	// float4 c = tex2D(texture, in.uv.s, 1 - in.uv.t);
	color = glm::vec4(c.x, c.y, c.z, c.w);
	// color = glm::pow(color, glm::vec4(1.f / 2.2f));
}

}
}
