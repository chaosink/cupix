#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
using namespace std;

#include "cupix.hpp"

#include <cuda_gl_interop.h>

namespace cupix {

namespace cu {

__constant__ __device__ int w, h;
__constant__ __device__ float time;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> texture;

__constant__ __device__ int n_triangle;
__constant__ __device__ unsigned char clear_color[4];

extern __device__
void VertexShader(VertexIn &in, VertexOut &out, glm::mat4 &mvp);

extern __device__
void FragmentShader(FragmentIn &in, glm::vec4 &color);

__global__
void Clear(unsigned char *frame_buf, float *depth_buf) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= w || y >= h) return;
	int i_thread = y * w + x;
	frame_buf[i_thread * 3 + 0] = clear_color[0];
	frame_buf[i_thread * 3 + 1] = clear_color[1];
	frame_buf[i_thread * 3 + 2] = clear_color[2];
	depth_buf[i_thread] = 0;
}

__global__
void NormalSpace(VertexIn *in, VertexOut *out, glm::mat4 *mvp) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= n_triangle * 3) return;

	VertexShader(in[x], out[x], *mvp);
}

__global__
void WindowSpace(VertexOut *v) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= n_triangle * 3) return;

	float w_inv = 1.f / v[x].position.w;
	glm::mat4 m;
	glm::vec3 p = v[x].position * w_inv;
	p.x = (p.x * 0.5f + 0.5f) * w;
	p.y = (p.y * 0.5f + 0.5f) * h;
	v[x].position.x = p.x;
	v[x].position.y = p.y;
	v[x].position.z = p.z;
}

__global__
void GetAABB(VertexOut *v, AABB *aabb) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= n_triangle) return;

	glm::vec2
		p1(v[x * 3 + 0].position.x, v[x * 3 + 0].position.y),
		p2(v[x * 3 + 1].position.x, v[x * 3 + 1].position.y),
		p3(v[x * 3 + 2].position.x, v[x * 3 + 2].position.y);
	glm::vec2
		v_min = glm::min(glm::min(p1, p2), p3),
		v_max = glm::max(glm::max(p1, p2), p3);
	glm::ivec2
		c0 = glm::ivec2(0, 0),
		c1 = glm::ivec2(w - 1, h - 1),
		iv_min = v_min + 0.5f,
		iv_max = v_max + 0.5f;

	iv_min = glm::clamp(iv_min, c0, c1);
	iv_max = glm::clamp(iv_max, c0, c1);

	aabb[x].v[0] = iv_min;
	aabb[x].v[1] = iv_max;
}

__device__
void Interpolate(VertexOut *v, FragmentIn *f, glm::vec3 e) {
	glm::vec3 d(
		e.x / v[0].position.w,
		e.y / v[1].position.w,
		e.z / v[2].position.w);
	f->depth = 1 / (d.x + d.y + d.z);
	f->normal = (
		v[0].normal * d.x +
		v[1].normal * d.y +
		v[2].normal * d.z) * f->depth;
	f->color = (
		v[0].color * d.x +
		v[1].color * d.y +
		v[2].color * d.z) * f->depth;
	f->uv = (
		v[0].uv * d.x +
		v[1].uv * d.y +
		v[2].uv * d.z) * f->depth;
	f->z =
		e.x * v[0].position.z +
		e.y * v[1].position.z +
		e.z * v[2].position.z;
}

__global__
void Rasterize(VertexOut *v, float *depth_buf, unsigned char* frame_buf, glm::ivec2 corner, glm::ivec2 dim) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= dim.x || y >= dim.y) return;
	x += corner.x;
	y += corner.y;
	if(x >= w || y >= h) return;
	int i_thread = y * w + x;

	// if(v[0].position.z >  1 || v[1].position.z >  1 || v[2].position.z >  1
	// || v[0].position.z < -1 || v[1].position.z < -1 || v[2].position.z < -1)
	// 	return;
	glm::vec2
		p0 = glm::vec2(v[0].position.x, v[0].position.y),
		p1 = glm::vec2(v[1].position.x, v[1].position.y),
		p2 = glm::vec2(v[2].position.x, v[2].position.y);

	glm::vec2 d01 = p1 - p0, d12 = p2 - p1, d20 = p0 - p2;
	float e0 = glm::dot(d12, glm::vec2(y + 0.5f - p1.y, p1.x - x - 0.5f));
	float e1 = glm::dot(d20, glm::vec2(y + 0.5f - p2.y, p2.x - x - 0.5f));
	float e2 = glm::dot(d01, glm::vec2(y + 0.5f - p0.y, p0.x - x - 0.5f));

	if(e0 >= 0 && e1 >= 0 && e2 >= 0) {
		FragmentIn fragment = {glm::ivec2(x, y)};
		float e = e0 + e1 + e2;
		Interpolate(v, &fragment, glm::vec3(e0, e1, e2) / e);
		if(fragment.z > 1 || fragment.z < -1) return;
		if(1 - fragment.z > depth_buf[i_thread]) {
			depth_buf[i_thread] = 1 - fragment.z;
			glm::vec4 color;
			FragmentShader(fragment, color);
			glm::ivec4 icolor = color * 255.f;
			icolor = glm::clamp(icolor, glm::ivec4(0), glm::ivec4(255));
			frame_buf[i_thread * 3 + 0] = icolor.r;
			frame_buf[i_thread * 3 + 1] = icolor.g;
			frame_buf[i_thread * 3 + 2] = icolor.b;
		}
	}
}

}



CUPix::CUPix(int window_w, int window_h, GLuint pbo, bool record = false)
	: window_w_(window_w), window_h_(window_h), record_(record) {
	frame_ = new unsigned char[window_w_ * window_h_ * 3];
	hipMalloc(&depth_buf_, sizeof(float) * window_w_ * window_h_);
	hipMalloc(&frame_buf_, sizeof(float) * window_w_ * window_h_);
	hipMalloc(&mvp_buf_, sizeof(glm::mat4));
	hipMemcpyToSymbol(HIP_SYMBOL(cu::w), &window_w_, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cu::h), &window_h_, sizeof(int));
	hipGraphicsGLRegisterBuffer(&pbo_resource_, pbo, cudaGraphicsMapFlagsNone);
}

CUPix::~CUPix() {
	delete[] frame_;
	hipFree(depth_buf_);
	hipFree(frame_buf_);
	hipFree(mvp_buf_);
	hipGraphicsUnregisterResource(pbo_resource_);
}

void CUPix::MapResources() {
	size_t size;
	hipGraphicsMapResources(1, &pbo_resource_, NULL);
	hipGraphicsResourceGetMappedPointer((void**)&pbo_ptr_, &size, pbo_resource_);
}

void CUPix::UnmapResources() {
	hipGraphicsUnmapResources(1, &pbo_resource_, NULL);
}

void CUPix::ClearColor(float r, float g, float b, float a) {
	glm::ivec4 color(r * 255.f, g * 255.f, b * 255.f, a * 255.f);
	color = glm::clamp(color, glm::ivec4(0), glm::ivec4(255));
	glm::u8vec4 clear_color = color;
	hipMemcpyToSymbol(HIP_SYMBOL(cu::clear_color), &clear_color, 4);
}

void CUPix::Clear() {
	cu::Clear<<<dim3((window_w_-1)/32+1, (window_h_-1)/32+1), dim3(32, 32)>>>(pbo_ptr_, depth_buf_);
}

void CUPix::Draw() {
	cu::NormalSpace<<<(n_triangle_*3-1)/32+1, 32>>>(vertex_in_, vertex_out_, mvp_buf_);
	cu::WindowSpace<<<(n_triangle_*3-1)/32+1, 32>>>(vertex_out_);
	cu::GetAABB<<<(n_triangle_-1)/32+1, 32>>>(vertex_out_, aabb_buf_);
	hipMemcpy(aabb_, aabb_buf_, sizeof(AABB) * n_triangle_, hipMemcpyDeviceToHost);
	for(int i = 0; i < n_triangle_; i++) {
		glm::ivec2 dim = aabb_[i].v[1] - aabb_[i].v[0] + 1;
		cu::Rasterize<<<dim3((dim.x-1)/4+1, (dim.y-1)/8+1), dim3(4, 8)>>>
			(vertex_out_ + i * 3, depth_buf_, pbo_ptr_, aabb_[i].v[0], dim);
	}
	if(record_)
		hipMemcpy(frame_, pbo_ptr_, window_w_ * window_h_ * 3, hipMemcpyDeviceToHost);
}

void CUPix::VertexData(int size, float *position, float *normal, float *uv) {
	n_vertex_ = size;
	n_triangle_ = n_vertex_ / 3;
	VertexIn v[n_vertex_];
	for(int i = 0; i < n_vertex_; i++) {
		v[i].position = glm::vec3(position[i * 3], position[i * 3 + 1], position[i * 3 + 2]);
		v[i].normal = glm::vec3(normal[i * 3], normal[i * 3 + 1], normal[i * 3 + 2]);
		v[i].color = glm::vec3(rand() / RAND_MAX, rand() / RAND_MAX, rand() / RAND_MAX);
		v[i].uv = glm::vec2(uv[i * 2], uv[i * 2 + 1]);
	}
	hipMalloc(&vertex_in_, sizeof(v));
	hipMemcpy(vertex_in_, v, sizeof(v), hipMemcpyHostToDevice);
	hipMalloc(&aabb_buf_, sizeof(AABB) * n_triangle_);
	aabb_ = new AABB[n_triangle_];
	hipMalloc(&vertex_out_, sizeof(VertexOut) * n_vertex_);
	hipMemcpyToSymbol(HIP_SYMBOL(cu::n_triangle), &n_triangle_, sizeof(int));
}

void CUPix::MVP(glm::mat4 &mvp) {
	hipMemcpy(mvp_buf_, &mvp, sizeof(glm::mat4), hipMemcpyHostToDevice);
}

void CUPix::Time(double time) {
	float t = time;
	hipMemcpyToSymbol(HIP_SYMBOL(cu::time), &t, sizeof(float));
}

void CUPix::Texture(unsigned char *d, int w, int h) {
	unsigned char data[w * h * 4];
	for(int i = 0; i < w * h; i++) {
		data[i * 4 + 0] = d[i * 3 + 0];
		data[i * 4 + 1] = d[i * 3 + 1];
		data[i * 4 + 2] = d[i * 3 + 2];
		data[i * 4 + 3] = 0;
	}
	size_t pitch;
	hipMallocPitch((void**)&texture_buf_, &pitch, w * 4, h);
	hipMemcpy2D(
		texture_buf_, pitch,
		data, w * 4,
		w * 4, h,
		hipMemcpyHostToDevice);

	cu::texture.normalized = true;
	cu::texture.sRGB = true;
	cu::texture.filterMode = hipFilterModeLinear;
	cu::texture.addressMode[0] = hipAddressModeWrap;
	cu::texture.addressMode[1] = hipAddressModeWrap;
	hipChannelFormatDesc desc =
		hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

	hipBindTexture2D(NULL, cu::texture, texture_buf_, desc, w, h, pitch);
}

}
